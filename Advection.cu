#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <stdio.h>
#include <cfloat>

#ifdef USE_HAPI
#include "hapi.h"
#endif

#define USE_CUB             0
#define USE_SHARED_MEM      1
#define SUB_BLOCK_SIZE      8
#define NUM_DIMS            3

#define FLAT_IDX(i,j,k)     (((k) * (block_size+2) + (j)) * (block_size+2) + (i))
#define FLAT_IDX4(d,i,j,k)  ((((d) * (block_size+2) + (k)) * (block_size+2) + (j)) * (block_size+2) + (i))
#define ERROR_IDX(i,j,k)    ((((k)-2) * (block_size-2) + ((j)-2)) * (block_size-2) + ((i)-2))

#define gpuSafe(retval)     gpuPrintError((retval), __FILE__, __LINE__)
#define gpuCheck()          gpuPrintError(hipGetLastError(), __FILE__, __LINE__)

inline void gpuPrintError(hipError_t err, const char *file, int line) {
  if (err != hipSuccess)
    fprintf(stderr,"CUDA Error: %s at %s:%d\n", hipGetErrorString(err), file, line);
}

void gpuHostAlloc(void** ptr, size_t size) { gpuSafe(hipHostMalloc(ptr, size)); }
void gpuHostFree(void* ptr) { gpuSafe(hipHostFree(ptr)); }
void gpuDeviceAlloc(void** ptr, size_t size) { gpuSafe(hipMalloc(ptr, size)); }
void gpuDeviceFree(void* ptr) { gpuSafe(hipFree(ptr)); }
void gpuStreamCreate(hipStream_t* stream_ptr) { gpuSafe(hipStreamCreate(stream_ptr)); }
void gpuStreamDestroy(hipStream_t stream) { gpuSafe(hipStreamDestroy(stream)); }

__device__ static float atomicMax(float* address, float val)
{
  int* address_as_i = (int*) address;
  int old = *address_as_i, assumed;
  do {
    assumed = old;
    old = ::atomicCAS(address_as_i, assumed,
        __float_as_int(::fmaxf(val, __int_as_float(assumed))));
  } while (assumed != old);
  return __int_as_float(old);
}

__global__ void computeKernel(float* u2, float* u, float dx, float dy, float dz, float dt, float apx, float apy, float apz, float anx, float any, float anz, int block_size) {
  float up[3];
  float un[3];

  int gx = blockDim.x * blockIdx.x + threadIdx.x;
  int gy = blockDim.y * blockIdx.y + threadIdx.y;
  int gz = blockDim.z * blockIdx.z + threadIdx.z;

#if USE_SHARED_MEM
  __shared__ float u_s[SUB_BLOCK_SIZE][SUB_BLOCK_SIZE][SUB_BLOCK_SIZE];

  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;

  // Read u into shared memory
  if ((gx < (block_size + 2)) && (gy < (block_size + 2)) && (gz < (block_size + 2))) {
    u_s[tx][ty][tz] = u[FLAT_IDX(gx,gy,gz)];
  }
  __syncthreads();
#endif

  if (((gx >= 1 && gx <= block_size) && (gy >= 1 && gy <= block_size)) && (gz >= 1 && gz <= block_size)) {
#if USE_SHARED_MEM
    up[0] = (((tx < SUB_BLOCK_SIZE-1) ? (u_s[tx+1][ty][tz]) : (u[FLAT_IDX(gx+1,gy,gz)])) - u[FLAT_IDX(gx,gy,gz)])/dx;
    un[0] = (u[FLAT_IDX(gx,gy,gz)] - ((tx > 0) ? (u_s[tx-1][ty][tz]) : (u[FLAT_IDX(gx-1,gy,gz)])))/dx;
    up[1] = (((ty < SUB_BLOCK_SIZE-1) ? (u_s[tx][ty+1][tz]) : (u[FLAT_IDX(gx,gy+1,gz)])) - u[FLAT_IDX(gx,gy,gz)])/dy;
    un[1] = (u[FLAT_IDX(gx,gy,gz)] - ((ty > 0) ? (u_s[tx][ty-1][tz]) : (u[FLAT_IDX(gx,gy-1,gz)])))/dy;
    up[2] = (((tz < SUB_BLOCK_SIZE-1) ? (u_s[tx][ty][tz+1]) : (u[FLAT_IDX(gx,gy,gz+1)])) - u[FLAT_IDX(gx,gy,gz)])/dz;
    un[2] = (u[FLAT_IDX(gx,gy,gz)] - ((tz > 0) ? (u_s[tx][ty][tz-1]) : (u[FLAT_IDX(gx,gy,gz-1)])))/dz;
#else
    up[0] = (u[FLAT_IDX(gx+1,gy,gz)] - u[FLAT_IDX(gx,gy,gz)])/dx;
    un[0] = (u[FLAT_IDX(gx,gy,gz)] - u[FLAT_IDX(gx-1,gy,gz)])/dx;
    up[1] = (u[FLAT_IDX(gx,gy+1,gz)] - u[FLAT_IDX(gx,gy,gz)])/dy;
    un[1] = (u[FLAT_IDX(gx,gy,gz)] - u[FLAT_IDX(gx,gy-1,gz)])/dy;
    up[2] = (u[FLAT_IDX(gx,gy,gz+1)] - u[FLAT_IDX(gx,gy,gz)])/dz;
    un[2] = (u[FLAT_IDX(gx,gy,gz)] - u[FLAT_IDX(gx,gy,gz-1)])/dz;
#endif

    u2[FLAT_IDX(gx,gy,gz)] = u[FLAT_IDX(gx,gy,gz)] - dt*(apx*un[0] + anx*up[0]) - dt*(apy*un[1] + any*up[1]) - dt*(apz*un[2] + anz*up[2]);
  }
}

__global__ void computeAddKernel(float* u, float* u2, float* u3, int block_size) {
  int gx = blockDim.x * blockIdx.x + threadIdx.x + 1;
  int gy = blockDim.y * blockIdx.y + threadIdx.y + 1;
  int gz = blockDim.z * blockIdx.z + threadIdx.z + 1;

  u[FLAT_IDX(gx,gy,gz)] = 0.5*(u2[FLAT_IDX(gx,gy,gz)] + u3[FLAT_IDX(gx,gy,gz)]);
}

void invokeComputeKernel(hipStream_t computeStream, float* u, float* d_u, float* d_u2, float* d_u3, float dx, float dy, float dz, float dt, float apx, float apy, float apz, float anx, float any, float anz, int block_size, void* cb) {
  // Copy u to GPU
  size_t u_size = sizeof(float)*(block_size+2)*(block_size+2)*(block_size+2);
  gpuSafe(hipMemcpyAsync(d_u, u, u_size, hipMemcpyHostToDevice, computeStream));
  gpuSafe(hipMemcpyAsync(d_u2, u, u_size, hipMemcpyHostToDevice, computeStream));
  gpuSafe(hipMemcpyAsync(d_u3, u, u_size, hipMemcpyHostToDevice, computeStream));

  // Execute first kernel to calculate u2
  int sub_block_cnt = ceil((float)(block_size+2)/SUB_BLOCK_SIZE);
  dim3 dimGrid(sub_block_cnt, sub_block_cnt, sub_block_cnt);
  dim3 dimBlock(SUB_BLOCK_SIZE, SUB_BLOCK_SIZE, SUB_BLOCK_SIZE);
  computeKernel<<<dimGrid, dimBlock, 0, computeStream>>>(d_u2, d_u, dx, dy, dz, dt, apx, apy, apz, anx, any, anz, block_size);
  gpuCheck();

  // Execute second kernel to calculate u3
  computeKernel<<<dimGrid, dimBlock, 0, computeStream>>>(d_u3, d_u2, dx, dy, dz, dt, apx, apy, apz, anx, any, anz, block_size);
  gpuCheck();

  // Execute last kernel to calculate new u
  sub_block_cnt = ceil((float)(block_size)/SUB_BLOCK_SIZE);
  dimGrid = dim3(sub_block_cnt, sub_block_cnt, sub_block_cnt);
  dimBlock = dim3(SUB_BLOCK_SIZE, SUB_BLOCK_SIZE, SUB_BLOCK_SIZE);
  computeAddKernel<<<dimGrid, dimBlock, 0, computeStream>>>(d_u, d_u2, d_u3, block_size);
  gpuCheck();

  // Copy new u back to host
  gpuSafe(hipMemcpyAsync(u, d_u, u_size, hipMemcpyDeviceToHost, computeStream));

#ifdef USE_HAPI
  // Use HAPI callback to get notified once the results are computed on the GPU
  hapiAddCallback(computeStream, cb);
#else
  // Wait until completion
  gpuSafe(hipStreamSynchronize(computeStream));
#endif
}

__global__ void decisionKernel1(float *u, float *delu, float *delua, float dx, float dy, float dz, int block_size) {
  float delx = 0.5/dx;
  float dely = 0.5/dy;
  float delz = 0.5/dz;

  int gx = blockDim.x * blockIdx.x + threadIdx.x;
  int gy = blockDim.y * blockIdx.y + threadIdx.y;
  int gz = blockDim.z * blockIdx.z + threadIdx.z;

#if USE_SHARED_MEM
  __shared__ float u_s[SUB_BLOCK_SIZE][SUB_BLOCK_SIZE][SUB_BLOCK_SIZE];

  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;

  // Read u into shared memory
  if ((gx < (block_size + 2)) && (gy < (block_size + 2)) && (gz < (block_size + 2))) {
    u_s[tx][ty][tz] = u[FLAT_IDX(gx,gy,gz)];
  }
  __syncthreads();
#endif

  // Calculate differentials
  float u_pos, u_neg;
  if (((gx >= 1 && gx <= block_size) && (gy >= 1 && gy <= block_size)) && (gz >= 1 && gz <= block_size)) {
    // d/dx
#if USE_SHARED_MEM
    u_pos = (tx < SUB_BLOCK_SIZE-1) ? (u_s[tx+1][ty][tz]) : (u[FLAT_IDX(gx+1,gy,gz)]);
    u_neg = (tx > 0) ? (u_s[tx-1][ty][tz]) : (u[FLAT_IDX(gx-1,gy,gz)]);
#else
    u_pos = u[FLAT_IDX(gx+1,gy,gz)];
    u_neg = u[FLAT_IDX(gx-1,gy,gz)];
#endif
    delu[FLAT_IDX4(0,gx,gy,gz)] = (u_pos - u_neg)*delx;
    delua[FLAT_IDX4(0,gx,gy,gz)] = (fabsf(u_pos) + fabsf(u_neg))*delx;

    // d/dy
#if USE_SHARED_MEM
    u_pos = (ty < SUB_BLOCK_SIZE-1) ? (u_s[tx][ty+1][tz]) : (u[FLAT_IDX(gx,gy+1,gz)]);
    u_neg = (ty > 0) ? (u_s[tx][ty-1][tz]) : (u[FLAT_IDX(gx,gy-1,gz)]);
#else
    u_pos = u[FLAT_IDX(gx,gy+1,gz)];
    u_neg = u[FLAT_IDX(gx,gy-1,gz)];
#endif
    delu[FLAT_IDX4(1,gx,gy,gz)] = (u_pos - u_neg)*dely;
    delua[FLAT_IDX4(1,gx,gy,gz)] = (fabsf(u_pos) + fabsf(u_neg))*dely;

    // d/dz
#if USE_SHARED_MEM
    u_pos = (tz < SUB_BLOCK_SIZE-1) ? (u_s[tx][ty][tz+1]) : (u[FLAT_IDX(gx,gy,gz+1)]);
    u_neg = (tz > 0) ? (u_s[tx][ty][tz-1]) : (u[FLAT_IDX(gx,gy,gz-1)]);
#else
    u_pos = u[FLAT_IDX(gx,gy,gz+1)];
    u_neg = u[FLAT_IDX(gx,gy,gz-1)];
#endif
    delu[FLAT_IDX4(2,gx,gy,gz)] = (u_pos - u_neg)*delz;
    delua[FLAT_IDX4(2,gx,gy,gz)] = (fabsf(u_pos) + fabsf(u_neg))*delz;
  }
}

__global__ void decisionKernel2(float *delu, float *delua, float *errors, float refine_filter, float dx, float dy, float dz, int block_size) {
  float delx = 0.5/dx;
  float dely = 0.5/dy;
  float delz = 0.5/dz;
  float delu_n[3][NUM_DIMS * NUM_DIMS];

  int gx = blockDim.x * blockIdx.x + threadIdx.x + 1;
  int gy = blockDim.y * blockIdx.y + threadIdx.y + 1;
  int gz = blockDim.z * blockIdx.z + threadIdx.z + 1;

#if USE_SHARED_MEM
  __shared__ float delu_s[NUM_DIMS][SUB_BLOCK_SIZE][SUB_BLOCK_SIZE][SUB_BLOCK_SIZE];
  __shared__ float delua_s[NUM_DIMS][SUB_BLOCK_SIZE][SUB_BLOCK_SIZE][SUB_BLOCK_SIZE];
#if !USE_CUB
  __shared__ float maxError;
#endif

  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;

  // Read delu & delua into shared memory
  if (gx <= block_size && gy <= block_size && gz <= block_size) {
    for (int d = 0; d < NUM_DIMS; d++) {
      delu_s[d][tx][ty][tz] = delu[FLAT_IDX4(d,gx,gy,gz)];
      delua_s[d][tx][ty][tz] = delua[FLAT_IDX4(d,gx,gy,gz)];
    }
  }
#if !USE_CUB
  if (tx == 0 && ty == 0 && tz == 0) {
    maxError = 0;
  }
#endif
  __syncthreads();
#endif // USE_SHARED_MEM

  // Calculate error per thread
  float delu_pos, delu_neg;
  float delua_pos, delua_neg;
  float num = 0., denom = 0.;
  float error = 0.0f;
  if ((gx > 1 && gx < block_size) && (gy > 1 && gy < block_size) && (gz > 1 && gz < block_size)) {
    for (int d = 0; d < NUM_DIMS; d++) {
#if USE_SHARED_MEM
      delu_pos = (tx < SUB_BLOCK_SIZE-1) ? (delu_s[d][tx+1][ty][tz]) : (delu[FLAT_IDX4(d,gx+1,gy,gz)]);
      delu_neg = (tx > 0) ? (delu_s[d][tx-1][ty][tz]) : (delu[FLAT_IDX4(d,gx-1,gy,gz)]);
      delua_pos = (tx < SUB_BLOCK_SIZE-1) ? (delua_s[d][tx+1][ty][tz]) : (delua[FLAT_IDX4(d,gx+1,gy,gz)]);
      delua_neg = (tx > 0) ? (delua_s[d][tx-1][ty][tz]) : (delua[FLAT_IDX4(d,gx-1,gy,gz)]);
#else
      delu_pos = delu[FLAT_IDX4(d,gx+1,gy,gz)];
      delu_neg = delu[FLAT_IDX4(d,gx-1,gy,gz)];
      delua_pos = delua[FLAT_IDX4(d,gx+1,gy,gz)];
      delua_neg = delua[FLAT_IDX4(d,gx-1,gy,gz)];
#endif
      delu_n[0][3*d+0] = (delu_pos - delu_neg)*delx;
      delu_n[1][3*d+0] = (fabsf(delu_pos) + fabsf(delu_neg))*delx;
      delu_n[2][3*d+0] = (delua_pos + delua_neg)*delx;

#if USE_SHARED_MEM
      delu_pos = (ty < SUB_BLOCK_SIZE-1) ? (delu_s[d][tx][ty+1][tz]) : (delu[FLAT_IDX4(d,gx,gy+1,gz)]);
      delu_neg = (ty > 0) ? (delu_s[d][tx][ty-1][tz]) : (delu[FLAT_IDX4(d,gx,gy-1,gz)]);
      delua_pos = (ty < SUB_BLOCK_SIZE-1) ? (delua_s[d][tx][ty+1][tz]) : (delua[FLAT_IDX4(d,gx,gy+1,gz)]);
      delua_neg = (ty > 0) ? (delua_s[d][tx][ty-1][tz]) : (delua[FLAT_IDX4(d,gx,gy-1,gz)]);
#else
      delu_pos = delu[FLAT_IDX4(d,gx,gy+1,gz)];
      delu_neg = delu[FLAT_IDX4(d,gx,gy-1,gz)];
      delua_pos = delua[FLAT_IDX4(d,gx,gy+1,gz)];
      delua_neg = delua[FLAT_IDX4(d,gx,gy-1,gz)];
#endif
      delu_n[0][3*d+1] = (delu_pos - delu_neg)*dely;
      delu_n[1][3*d+1] = (fabsf(delu_pos) + fabsf(delu_neg))*dely;
      delu_n[2][3*d+1] = (delua_pos + delua_neg)*dely;

#if USE_SHARED_MEM
      delu_pos = (tz < SUB_BLOCK_SIZE-1) ? (delu_s[d][tx][ty][tz+1]) : (delu[FLAT_IDX4(d,gx,gy,gz+1)]);
      delu_neg = (tz > 0) ? (delu_s[d][tx][ty][tz-1]) : (delu[FLAT_IDX4(d,gx,gy,gz-1)]);
      delua_pos = (tz < SUB_BLOCK_SIZE-1) ? (delua_s[d][tx][ty][tz+1]) : (delua[FLAT_IDX4(d,gx,gy,gz+1)]);
      delua_neg = (tz > 0) ? (delua_s[d][tx][ty][tz-1]) : (delua[FLAT_IDX4(d,gx,gy,gz-1)]);
#else
      delu_pos = delu[FLAT_IDX4(d,gx,gy,gz+1)];
      delu_neg = delu[FLAT_IDX4(d,gx,gy,gz-1)];
      delua_pos = delua[FLAT_IDX4(d,gx,gy,gz+1)];
      delua_neg = delua[FLAT_IDX4(d,gx,gy,gz-1)];
#endif
      delu_n[0][3*d+2] = (delu_pos - delu_neg)*delz;
      delu_n[1][3*d+2] = (fabsf(delu_pos) + fabsf(delu_neg))*delz;
      delu_n[2][3*d+2] = (delua_pos + delua_neg)*delz;
    }

    for (int dd = 0; dd < NUM_DIMS * NUM_DIMS; dd++) {
      num = num + pow(delu_n[0][dd], 2.);
      denom = denom + pow(delu_n[1][dd], 2.) + (refine_filter * delu_n[2][dd]) * 2;
    }

    if (denom == 0. && num != 0.) {
      error = FLT_MAX;
    }
    else if (denom != 0.0) {
      error = num/denom;
    }

#if USE_CUB
    // Store error in global memory
    errors[ERROR_IDX(gx,gy,gz)] = error;
#else
#if USE_SHARED_MEM
    atomicMax(&maxError, error);
#else
    atomicMax(errors, error);
#endif // USE_SHARED_MEM
#endif // USE_CUB
  }

#if !USE_CUB
#if USE_SHARED_MEM
  __syncthreads();
  if (tx == 0 && ty == 0 && tz == 0)
    atomicMax(errors, maxError);
#endif // USE_SHARED_MEM
#endif // USE_CUB
}

float invokeDecisionKernel(hipStream_t decisionStream, float* u, float* h_error, float* d_error, float* d_u, float* d_delu, float* d_delua, float refine_filter, float dx, float dy, float dz, int block_size, void* cb) {
  // Find the maximum error value, which will be used to decide whether to refine
#if USE_CUB
  size_t errors_size = sizeof(float)*(block_size-2)*(block_size-2)*(block_size-2);
  float *d_errors;
  gpuSafe(hipMalloc(&d_errors, errors_size));
#endif

  // Intiailize memory on device
  size_t u_size = sizeof(float)*(block_size+2)*(block_size+2)*(block_size+2);
  size_t delu_size = NUM_DIMS * u_size;
  *h_error = 0.0f;
  gpuSafe(hipMemset(d_delu, 0, delu_size));
  gpuSafe(hipMemset(d_delua, 0, delu_size));
  gpuSafe(hipMemset(d_error, 0, sizeof(float)));

  // Copy u to device
  gpuSafe(hipMemcpyAsync(d_u, u, u_size, hipMemcpyHostToDevice, decisionStream));

  // Execute first kernel to calculate delu and delua
  int sub_block_cnt = ceil((float)(block_size+2)/SUB_BLOCK_SIZE);
  dim3 dimGrid(sub_block_cnt, sub_block_cnt, sub_block_cnt);
  dim3 dimBlock(SUB_BLOCK_SIZE, SUB_BLOCK_SIZE, SUB_BLOCK_SIZE);
  decisionKernel1<<<dimGrid, dimBlock, 0, decisionStream>>>(d_u, d_delu, d_delua, dx, dy, dz, block_size);
  gpuCheck();

  // Execute second kernel to calculate errors
  sub_block_cnt = ceil((float)block_size/SUB_BLOCK_SIZE);
  dimGrid = dim3(sub_block_cnt, sub_block_cnt, sub_block_cnt);
#if USE_CUB
  decisionKernel2<<<dimGrid, dimBlock, 0, decisionStream>>>(d_delu, d_delua, d_errors, refine_filter, dx, dy, dz, block_size);
  gpuCheck();

  // Max reduction using cub (TODO: can multiple instances of this run concurrently?)
  void *d_temp_storage = NULL;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_errors, d_error, (block_size-2)*(block_size-2)*(block_size-2));
  hipMalloc(&d_temp_storage, temp_storage_bytes);
  hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_errors, d_error, (block_size-2)*(block_size-2)*(block_size-2));
  gpuSafe(hipMemcpyAsync(h_error, d_error, sizeof(float), hipMemcpyDeviceToHost, decisionStream));

  // Wait until completion
  gpuSafe(hipDeviceSynchronize());

  // Deallocate memory
  gpuSafe(hipFree(d_errors));
  gpuSafe(hipFree(d_temp_storage));
#else
  decisionKernel2<<<dimGrid, dimBlock, 0, decisionStream>>>(d_delu, d_delua, d_error, refine_filter, dx, dy, dz, block_size);
  gpuCheck();

  gpuSafe(hipMemcpyAsync(h_error, d_error, sizeof(float), hipMemcpyDeviceToHost, decisionStream));


// TODO Don't use HAPI version because it sometimes results in different refinement decisions
//#ifdef USE_HAPI
#if 0
  // Use HAPI callback to get notified once the results are computed on the GPU
  hapiAddCallback(decisionStream, cb);
#else
  // Wait until completion
  gpuSafe(hipStreamSynchronize(decisionStream));
#endif
#endif // USE_CUB

//#ifdef USE_HAPI
#if 0
  // Just return a dummy value, we will be notified once the actual result is computed
  return 0;
#else
  // Return maximum error
  return (*h_error);
#endif
}
